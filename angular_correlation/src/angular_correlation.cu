#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>

#include<hip/hip_runtime.h>

using namespace std;

#define SUBMATRIX_SIZE 16384

////////////////////////////////////////////////////////////////////////
// Number of histogram bins has to be edited by hand, prior to
// copmilation.
////////////////////////////////////////////////////////////////////////

#define DEFAULT_NBINS 254 
//#define DEFAULT_NBINS 126 
//#define DEFAULT_NBINS 62 
//#define DEFAULT_NBINS 30 

#define CONV_FACTOR 57.2957795 // 180/pi


int doCalcRaDec(FILE *infile0, FILE *infile1, FILE *outfile, bool silent_on_GPU_testing, float scale_factor, int nbins, float hist_lower_range, float hist_upper_range, float hist_bin_width, int log_binning_flag, bool two_different_files, float conv_factor_angle);

int doCalcMpc(FILE *infile0, FILE *infile1, FILE *outfile, bool silent_on_GPU_testing, float scale_factor, int nbins, float hist_lower_range, float hist_upper_range, float hist_bin_width, int log_binning_flag, bool two_different_files, float conv_factor_angle);

void getDeviceDiagnostics(int tot_Gals, int n_coords);

////////////////////////////////////////////////////////////////////////
// Kernel to calculate angular distances between galaxies and histogram
// the distances.
////////////////////////////////////////////////////////////////////////
__global__ void distance(volatile float *a0, volatile float *d0, volatile float *a1, volatile float *d1, int xind, int yind, int max_xind, int max_yind, volatile int *dev_hist, float hist_min, float hist_max, int nbins, float bin_width, int log_binning=0, bool two_different_files=1, float conv_factor_angle=57.2957795)
{

    ////////////////////////////////////////////////////////////////////////////
    // Idx will keep track of which thread is being calculated within a given 
    // warp.
    ////////////////////////////////////////////////////////////////////////////
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // This should range to SUBMATRIX_SIZE

    idx += xind;

    ////////////////////////////////////////////////////////////////////////
    // Shared memory stuff.
    ////////////////////////////////////////////////////////////////////////
    __shared__ int shared_hist[DEFAULT_NBINS+2];
    // Note that we only clear things out for the first thread on each block.
    if(threadIdx.x==0)
    {
        for (int i=0;i<nbins+2;i++)
            shared_hist[i] = 0;
    }
    __syncthreads();
    ////////////////////////////////////////////////////////////////////////

    if (idx<max_xind)
    {
        int i=0;

        float alpha_rad = a0[idx];
        float delta0 = d0[idx];
        float cos_d0 = cos(delta0);
        float sin_d0 = sin(delta0);
        float dist;

        int bin_index = 0; 

        float a_diff, sin_a_diff, cos_a_diff;
        float cos_d1, sin_d1, numer, denom, mult1, mult2;    
        float d1_rad;

        bool do_calc = 1;

        int ymax = yind + SUBMATRIX_SIZE;

        if (ymax>max_yind)
        {
            ymax = max_yind;
        }

        for(i=yind; i<ymax; i++)
        {
            if (two_different_files)
            {
                do_calc = 1;
            }
            else // Doing the same file
            {
                if(idx > i)
                    do_calc=1;
                else
                    do_calc=0;
            }
            //if(idx > i) ///////// CHECK THIS
            if (do_calc)
            {
                a_diff = a1[i] - alpha_rad;
                d1_rad = d1[i];

                sin_a_diff = sin(a_diff);
                cos_a_diff = cos(a_diff);

                sin_d1 = sin(d1_rad);
                cos_d1 = cos(d1_rad);

                mult1 = cos_d1 * cos_d1 * sin_a_diff * sin_a_diff;
                mult2 = cos_d0 * sin_d1 - sin_d0 * cos_d1 * cos_a_diff;
                mult2 = mult2 * mult2;

                numer = sqrt(mult1 + mult2); 

                denom = sin_d0 *sin_d1 + cos_d0 * cos_d1 * cos_a_diff;

                dist = atan2(numer,denom);  
                dist *= conv_factor_angle;  // Convert to degrees or what have you.

                if(dist < hist_min)
                    bin_index = 0; 
                else if(dist >= hist_max)
                    bin_index = nbins + 1;
                else
                {
                    if (log_binning==0)
                    {
                        bin_index = int((dist-hist_min)/bin_width) + 1;
                    }
                    else if (log_binning==1)// log binning
                    {
                        bin_index = int((log(dist)-log(hist_min))/bin_width) + 1;
                    }
                    else if (log_binning==2)// log 10 binning
                    {
                        bin_index = int((log10(dist)-log10(hist_min))/bin_width) + 1;
                    }
                }

                atomicAdd(&shared_hist[bin_index],1);

            }
        }
    }

    __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<nbins+2;i++)
            dev_hist[i+(blockIdx.x*(nbins+2))]=shared_hist[i];
    }

}





////////////////////////////////////////////////////////////////////////
// Kernel to calculate angular distances between galaxies and histogram
// the distances.
// Assuming coordinates are already in x,y,z (in Mpc)
////////////////////////////////////////////////////////////////////////
__global__ void distanceMpc(volatile float *x0, volatile float *y0,  volatile float *z0,  volatile float *x1, volatile float *y1, volatile float *z1, int xind, int yind, int max_xind, int max_yind, volatile int *dev_hist, float hist_min, float hist_max, int nbins, float bin_width, int log_binning=0, bool two_different_files=1, float conv_factor_angle=57.2957795)
{

    ////////////////////////////////////////////////////////////////////////////
    // Idx will keep track of which thread is being calculated within a given 
    // warp.
    ////////////////////////////////////////////////////////////////////////////
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // This should range to SUBMATRIX_SIZE

    idx += xind;

    ////////////////////////////////////////////////////////////////////////
    // Shared memory stuff.
    ////////////////////////////////////////////////////////////////////////
    __shared__ int shared_hist[DEFAULT_NBINS+2];
    // Note that we only clear things out for the first thread on each block.
    if(threadIdx.x==0)
    {
        for (int i=0;i<nbins+2;i++)
            shared_hist[i] = 0;
    }
    __syncthreads();
    ////////////////////////////////////////////////////////////////////////

    if (idx<max_xind)
    {
        int i=0;

        float dist, xdiff, ydiff, zdiff;

        int bin_index = 0; 

        
        bool do_calc = 1;

        int ymax = yind + SUBMATRIX_SIZE;

        if (ymax>max_yind)
        {
            ymax = max_yind;
        }

        for(i=yind; i<ymax; i++)
        {
            if (two_different_files)
            {
                do_calc = 1;
            }
            else // Doing the same file
            {
                if(idx > i)
                    do_calc=1;
                else
                    do_calc=0;
            }
            //if(idx > i) ///////// CHECK THIS
            if (do_calc)
            {
	        // this is a way simpler calculation. We already have the x,y,z coodis in co-moving distance, so we can simply do the distance

		xdiff = x0[idx] - x1[i];
		ydiff = y0[idx] - y1[i];
		zdiff = z0[idx] - z1[i];

		dist = sqrt( (xdiff*xdiff) + (ydiff*ydiff) + (zdiff*zdiff));    
		//dist = (ydiff*ydiff);

                if(dist < hist_min)
                    bin_index = 0; 
                else if(dist >= hist_max)
                    bin_index = nbins + 1;
                else
                {
                    if (log_binning==0)
                    {
                        bin_index = int((dist-hist_min)/bin_width) + 1;
                    }
                    else if (log_binning==1)// log binning
                    {
                        bin_index = int((log(dist)-log(hist_min))/bin_width) + 1;
                    }
                    else if (log_binning==2)// log 10 binning
                    {
                        bin_index = int((log10(dist)-log10(hist_min))/bin_width) + 1;
                    }
                }

                atomicAdd(&shared_hist[bin_index],1);
                //atomicAdd(&shared_hist[9],1);
                /*
                if (int(dist)<hist_max)
                    atomicAdd(&shared_hist[int(dist)],1);
                    */

            }
        }
    }

    __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<nbins+2;i++)
            dev_hist[i+(blockIdx.x*(nbins+2))]=shared_hist[i];
    }

}



////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
// Main
////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // Needed for parsing command-line arguments.
    extern char *optarg;
    extern int optind, optopt, opterr;
    int c;
    char *outfilename = NULL;
    char defaultoutfilename[256];
    sprintf(defaultoutfilename,"default_out.dat");

    float hist_lower_range = 0.0000001;
    float hist_upper_range = 0;
    int nbins = DEFAULT_NBINS;
    float hist_bin_width = 0.05;
    int log_binning_flag = 0; // False

    float scale_factor = 1.0; // For if we need to convert input to arcsec or arcmin
    float conv_factor_angle = 57.2957795; // 180/pi // For if we need to convert arcdistance to arcsec or arcmin
    int radec_input = 1; // are we using ra/dec coords, or x/y/z coords? 
    bool silent_on_GPU_testing = false;
    int cuda_device = 0;

    ////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////

    while ((c = getopt(argc, argv, "ao:L:l:w:smSd:p")) != -1) {
        switch(c) {
            case 'L':
                printf("L is set\n");
                hist_lower_range = atof(optarg);
                break;
            case 'w':
                hist_bin_width = atof(optarg);
                printf("Histogram bin width: %f\n",hist_bin_width);
                break;
            case 'l':
                log_binning_flag = atoi(optarg);
                printf("Will use log binning.\n");
                break;
            case 's':
                scale_factor = 206264.0; // To convert arcseconds to radians.
                conv_factor_angle *= 3600.0; // convert radians to arcseconds.
                printf("Reading in values assuming they are arcseconds.\n");
                printf("scale_factor: %f\n",scale_factor);
                printf("conv_factor_angle: %f\n",conv_factor_angle);
                break;
            case 'm':
                scale_factor = 3437.74677; // To convert arcminutes to radians.
                conv_factor_angle *= 60.0; // convert radians to arcminutes.
                printf("scale_factor: %f\n",scale_factor);
                printf("conv_factor_angle: %f\n",conv_factor_angle);
                printf("Reading in values assuming they are arcminutes.\n");
                break;
            case 'o':
                outfilename = optarg;
                printf("Output filename is %s\n", outfilename);
                break;
            case 'd':
                cuda_device = atoi(optarg); // Use this CUDA device.
                printf("Will attempt to use CUDA device %d\n",cuda_device);
                break;
            case 'S':
                printf("Silent mode - don't run the GPU test (suppresses some output)\n");
                silent_on_GPU_testing = true;
                break;
            case 'p':
                printf("Using input files in Mpc format");
                radec_input = 0;
                break;
            case '?':
                printf("unknown arg %c\n", optopt);
                break;
        }
    }

    if (argc < 2)
    {

        printf("\nMust pass in at least two input files on command line!\n");
        printf("\nUsage: ", argv[0] );
        //printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    // Set a default output file name, if none was passed in on the 
    // command line.
    if (outfilename == NULL) 
    {
        outfilename = defaultoutfilename;
        printf("Output filename is %s\n", outfilename);
    }

    float temp_lo = hist_lower_range;
    if (hist_upper_range == 0)
    {
        if (log_binning_flag==0)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = temp_lo + hist_bin_width;
                temp_lo = hist_upper_range;
            }
        }
        else if (log_binning_flag==1)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = exp(log(temp_lo) + hist_bin_width);
                temp_lo = hist_upper_range;
            }
        }
        else if (log_binning_flag==2)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = pow(10,(log10(temp_lo) + hist_bin_width));
                temp_lo = hist_upper_range;
            }
        }
    }
    printf("hist_upper_range: %f\n",hist_upper_range);

    FILE *infile0, *infile1, *outfile ;
    infile0 = fopen(argv[optind],"r");
    infile1 = fopen(argv[optind+1],"r");

    printf("Opening input file 0: %s\n",argv[optind]);
    printf("Opening input file 1: %s\n",argv[optind+1]);
    outfile = fopen(outfilename, "w");

    ////////////////////////////////////////////////////////////////////////////
    // Check to see if the two files are actually the same file.
    // This is the case for the DD and RR calculations and change slightly
    // the exact calculations being performed.
    ////////////////////////////////////////////////////////////////////////////
    bool two_different_files = 1;
    if (strcmp(argv[optind],argv[optind+1])==0)
    {
        two_different_files = 0;
        printf("Using the same file!\n");
    }
    printf("\n");

    ////////////////////////////////////////////////////////////////////////
    // Set the CUDA device. This is useful if your machine has multiple GPUs
    // on it. 
    ////////////////////////////////////////////////////////////////////////
    hipError_t error_id = hipSetDevice(cuda_device);
    if (error_id == hipSuccess) {
        printf( "hipSetDevice returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
    }
    else{
        printf( "hipSetDevice failed on Device %d!\n\n",cuda_device);
        exit(-1);
    }

    if(radec_input==1) int success = doCalcRaDec(infile0, infile1, outfile, silent_on_GPU_testing, scale_factor, nbins, hist_lower_range, hist_upper_range, hist_bin_width, log_binning_flag, two_different_files, conv_factor_angle);

    else  int success = doCalcMpc(infile0, infile1, outfile, silent_on_GPU_testing, scale_factor, nbins, hist_lower_range, hist_upper_range, hist_bin_width, log_binning_flag, two_different_files, conv_factor_angle);

}






////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
// Module that runs the calculations for input files in the format ra:dec
////////////////////////////////////////////////////////////////////////

int doCalcRaDec(FILE *infile0, FILE *infile1, FILE *outfile, bool silent_on_GPU_testing, float scale_factor, int nbins, float hist_lower_range, float hist_upper_range, float hist_bin_width, int log_binning_flag, bool two_different_files, float conv_factor_angle){
    
    float *d_alpha0, *d_delta0;
    float *h_alpha0, *h_delta0;

    float *d_alpha1, *d_delta1;
    float *h_alpha1, *h_delta1;

    int NUM_GALAXIES0;
    int NUM_GALAXIES1;

    //////////////////////////////////////////////////////////////////////
    // Read in the galaxy files.
    ////////////////////////////////////////////////////////////////////////////
    // Read in the first file
    ////////////////////////////////////////////////////////////////////////////

    fscanf(infile0, "%d", &NUM_GALAXIES0);

    int size_of_galaxy_array0 = NUM_GALAXIES0 * sizeof(float);    
    printf("SIZE 0 # GALAXIES: %d\n",NUM_GALAXIES0);

    h_alpha0 = (float*)malloc(size_of_galaxy_array0);
    h_delta0 = (float*)malloc(size_of_galaxy_array0);
    float temp0, temp1;

    for(int i=0; i<NUM_GALAXIES0; i++)
    {
        fscanf(infile0, "%f %f", &temp0, &temp1);
        h_alpha0[i] = temp0/scale_factor;
        h_delta0[i] = temp1/scale_factor;
        //if (i<10)
        //printf("%e %e\n", h_alpha0[i], h_delta0[i]);
    }

    ////////////////////////////////////////////////////////////////////////////
    // Read in the second file
    ////////////////////////////////////////////////////////////////////////////

    fscanf(infile1, "%d", &NUM_GALAXIES1);

    int size_of_galaxy_array1 = NUM_GALAXIES1 * sizeof(float);    
    printf("SIZE 1 # GALAXIES: %d\n",NUM_GALAXIES1);

    h_alpha1 = (float*)malloc(size_of_galaxy_array1);
    h_delta1 = (float*)malloc(size_of_galaxy_array1);

    for(int i=0; i<NUM_GALAXIES1; i++)
    {
        fscanf(infile1, "%f %f", &temp0, &temp1);
        h_alpha1[i] = temp0/scale_factor;
        h_delta1[i] = temp1/scale_factor;
        //if (i<10)
        //printf("%e %e\n", h_alpha1[i], h_delta1[i]);
    }

//get device diagnostics
 if (!silent_on_GPU_testing) getDeviceDiagnostics(NUM_GALAXIES0+NUM_GALAXIES1, 2);

    ////////////////////////////////////////////////////////////////////////////
    // Allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;

    int size_hist = SUBMATRIX_SIZE * (nbins+2);
    int size_hist_bytes = size_hist*sizeof(int);

    hist = (int*)malloc(size_hist_bytes);
    memset(hist, 0, size_hist_bytes);

    printf("Size of histogram: %d bytes\n",size_hist_bytes);
    hipMalloc((void **) &dev_hist, (size_hist_bytes));
    hipMemset(dev_hist, 0, size_hist_bytes);

    unsigned long  *hist_array;

    int hist_array_size = (nbins+2) * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
    printf("Size of histogram array: %d bytes\n",hist_array_size);
    memset(hist_array,0,hist_array_size); 

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    // 128*4 = 512, the amount of memory needed for one histogram.
    // 8192*4 = 32768 is max memory to ask for for the histograms.
    // 8192/128 = 64, is is the right number of blocks?
    grid.x = 8192/(DEFAULT_NBINS+2); // Is this the number of blocks?
    block.x = SUBMATRIX_SIZE/grid.x; // Is this the number of threads per block? NUM_GALAXIES/block.x;
    // SUBMATRIX is the number of threads per warp? Per kernel call?
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &d_alpha0, size_of_galaxy_array0 );
    hipMalloc((void **) &d_delta0, size_of_galaxy_array0 );

    hipMalloc((void **) &d_alpha1, size_of_galaxy_array1 );
    hipMalloc((void **) &d_delta1, size_of_galaxy_array1 );

    // Check to see if we allocated enough memory.
    if (0==d_alpha0 || 0==d_delta0 || 0==d_alpha1 || 0==d_delta1 || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    // Initialize array to all 0's
    hipMemset(d_alpha0,0,size_of_galaxy_array0);
    hipMemset(d_delta0,0,size_of_galaxy_array0);
    hipMemset(d_alpha1,0,size_of_galaxy_array1);
    hipMemset(d_delta1,0,size_of_galaxy_array1);

    hipMemcpy(d_alpha0, h_alpha0, size_of_galaxy_array0, hipMemcpyHostToDevice );
    hipMemcpy(d_delta0, h_delta0, size_of_galaxy_array0, hipMemcpyHostToDevice );
    hipMemcpy(d_alpha1, h_alpha1, size_of_galaxy_array1, hipMemcpyHostToDevice );
    hipMemcpy(d_delta1, h_delta1, size_of_galaxy_array1, hipMemcpyHostToDevice );

    int x, y;
    int num_submatrices_x = NUM_GALAXIES0 / SUBMATRIX_SIZE;
    int num_submatrices_y = NUM_GALAXIES1 / SUBMATRIX_SIZE;
    // Take care of edges of matrix.
    if (NUM_GALAXIES0%SUBMATRIX_SIZE != 0)
    {
        num_submatrices_x += 1;
    }
    if (NUM_GALAXIES1%SUBMATRIX_SIZE != 0)
    {
        num_submatrices_y += 1;
    }


    printf("Breaking down the calculations.\n");
    printf("Number of submatrices: %dx%d\n",num_submatrices_x,num_submatrices_y);
    printf("Number of calculations per submatrices: %dx%d\n",SUBMATRIX_SIZE,SUBMATRIX_SIZE);

    int bin_index = 0;
    for(int k = 0; k < num_submatrices_y; k++)
    {
        y = k*SUBMATRIX_SIZE;
        //printf("%d %d\n",k,y);
        for(int j = 0; j < num_submatrices_x; j++)
        {
            x = j*SUBMATRIX_SIZE; 

            //printf("----\n");
            //printf("%d %d\t\t%d %d\n",k,y,j,x);
            //printf("----\n");

            // Set the histogram to all zeros each time.
            hipMemset(dev_hist,0,size_hist_bytes);

            int max_x = NUM_GALAXIES0;
            int max_y = NUM_GALAXIES1;

            distance<<<grid,block>>>(d_alpha0, d_delta0,d_alpha1, d_delta1, x, y, max_x, max_y, dev_hist, hist_lower_range, hist_upper_range, nbins, hist_bin_width, log_binning_flag, two_different_files,conv_factor_angle);
            hipMemcpy(hist, dev_hist, size_hist_bytes, hipMemcpyDeviceToHost);

            ////////////////////////////////////////////////////////////////////
            // Sum up the histograms from each thread (hist).
            ////////////////////////////////////////////////////////////////////
            for(int m=0; m<size_hist; m++)
            {
                bin_index = m%(nbins+2);
                hist_array[bin_index] += hist[m];
            }    
        }  
    }

    unsigned long total = 0;

    float lo = hist_lower_range;
    float hi = 0;
    for(int k=0; k<nbins+1; k++)
    {
        if (k==0)
        {
            //fprintf(outfile, "Underflow below %.3e %s %lu \n", lo, ",",  hist_array[k]);
        }
        else
        {
            if (log_binning_flag==0)
            {
                hi = lo + hist_bin_width;
            }
            else if (log_binning_flag==1)
            {
                //printf("lo: %f\t\tlog(lo): %f\n",lo,log(lo));
                hi = exp(log(lo) + hist_bin_width);
            }
            else if (log_binning_flag==2)
            {
                //printf("lo: %f\t\tlog10(lo): %f\n",lo,log10(lo));
                hi = pow(10,(log10(lo) + hist_bin_width));
            }

            fprintf(outfile, "%.3e %.3e %lu \n",lo,hi,hist_array[k]);
            total += hist_array[k];

            lo = hi;
        }
    }
    printf("total: %lu \n", total);

    fclose(infile0);
    fclose(infile1);
    fclose(outfile);

    free(h_alpha0);
    free(h_delta0);
    free(h_alpha1);
    free(h_delta1);
    free(hist);

    hipFree(d_alpha0);
    hipFree(d_delta0);  
    hipFree(d_alpha1);
    hipFree(d_delta1);  
    hipFree(dev_hist);

    return 0;
}  
//////////////////////////////////////////////////////////////////////





////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
// Module that runs the calculations for input files in the format x:y:z in Mpc
////////////////////////////////////////////////////////////////////////

int doCalcMpc(FILE *infile0, FILE *infile1, FILE *outfile, bool silent_on_GPU_testing, float scale_factor, int nbins, float hist_lower_range, float hist_upper_range, float hist_bin_width, int log_binning_flag, bool two_different_files, float conv_factor_angle){




    float *d_x0, *d_y0, *d_z0;
    float *h_x0, *h_y0, *h_z0;

    float *d_x1, *d_y1, *d_z1;
    float *h_x1, *h_y1, *h_z1;

    int NUM_GALAXIES0;
    int NUM_GALAXIES1;

    //////////////////////////////////////////////////////////////////////
    // Read in the galaxy files.
    ////////////////////////////////////////////////////////////////////////////
    // Read in the first file
    ////////////////////////////////////////////////////////////////////////////

    fscanf(infile0, "%d", &NUM_GALAXIES0);

    int size_of_galaxy_array0 = NUM_GALAXIES0 * sizeof(float);    
    printf("SIZE 0 # GALAXIES: %d\n",NUM_GALAXIES0);

    h_x0 = (float*)malloc(size_of_galaxy_array0);
    h_y0 = (float*)malloc(size_of_galaxy_array0);
    h_z0 = (float*)malloc(size_of_galaxy_array0);
    float temp0, temp1, temp2;

    for(int i=0; i<NUM_GALAXIES0; i++)
    {
        fscanf(infile0, "%f %f %f", &temp0, &temp1, &temp2);
        h_x0[i] = temp0/scale_factor;
        h_y0[i] = temp1/scale_factor;
        h_z0[i] = temp2/scale_factor;
        if (i<10)
            printf("%f %f %f\n", h_x0[i], h_y0[i], h_z0[i]);
    }


    ////////////////////////////////////////////////////////////////////////////
    // Read in the second file
    ////////////////////////////////////////////////////////////////////////////

    fscanf(infile1, "%d", &NUM_GALAXIES1);

    int size_of_galaxy_array1 = NUM_GALAXIES1 * sizeof(float);    
    printf("SIZE 1 # GALAXIES: %d\n",NUM_GALAXIES1);

    h_x1 = (float*)malloc(size_of_galaxy_array1);
    h_y1 = (float*)malloc(size_of_galaxy_array1);
    h_z1 = (float*)malloc(size_of_galaxy_array1);

    for(int i=0; i<NUM_GALAXIES1; i++)
    {
        fscanf(infile1, "%f %f %f", &temp0, &temp1, &temp2);
        h_x1[i] = temp0/scale_factor;
        h_y1[i] = temp1/scale_factor;
        h_z1[i] = temp2/scale_factor;
        if (i<10)
            printf("%f %f %f\n", h_x1[i], h_y1[i], h_z1[i]);
    }

// get device diagnostics 
 if (!silent_on_GPU_testing) getDeviceDiagnostics(NUM_GALAXIES0+NUM_GALAXIES1, 2);


    ////////////////////////////////////////////////////////////////////////////
    // Allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;

    int size_hist = SUBMATRIX_SIZE * (nbins+2);
    int size_hist_bytes = size_hist*sizeof(int);

    hist = (int*)malloc(size_hist_bytes);
    memset(hist, 0, size_hist_bytes);

    printf("Size of histogram: %d bytes\n",size_hist_bytes);
    hipMalloc((void **) &dev_hist, (size_hist_bytes));
    hipMemset(dev_hist, 0, size_hist_bytes);

    unsigned long  *hist_array;

    int hist_array_size = (nbins+2) * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
    printf("Size of histogram array: %d bytes\n",hist_array_size);
    memset(hist_array,0,hist_array_size); 

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    // 128*4 = 512, the amount of memory needed for one histogram.
    // 8192*4 = 32768 is max memory to ask for for the histograms.
    // 8192/128 = 64, is is the right number of blocks?
    grid.x = 8192/(DEFAULT_NBINS+2); // Is this the number of blocks?
    block.x = SUBMATRIX_SIZE/grid.x; // Is this the number of threads per block? NUM_GALAXIES/block.x;
    // SUBMATRIX is the number of threads per warp? Per kernel call?
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &d_x0, size_of_galaxy_array0 );
    hipMalloc((void **) &d_y0, size_of_galaxy_array0 );
    hipMalloc((void **) &d_z0, size_of_galaxy_array0 );

    hipMalloc((void **) &d_x1, size_of_galaxy_array1 );
    hipMalloc((void **) &d_y1, size_of_galaxy_array1 );
    hipMalloc((void **) &d_z1, size_of_galaxy_array1 );

    // Check to see if we allocated enough memory.
    if (0==d_x0 || 0==d_y0 || 0==d_z0 || 0==d_x1 || 0==d_y1 || 0==d_z1 || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    // Initialize array to all 0's
    hipMemset(d_x0,0,size_of_galaxy_array0);
    hipMemset(d_y0,0,size_of_galaxy_array0);
    hipMemset(d_z0,0,size_of_galaxy_array0);
    hipMemset(d_x1,0,size_of_galaxy_array1);
    hipMemset(d_y1,0,size_of_galaxy_array1);
    hipMemset(d_z1,0,size_of_galaxy_array1);

    hipMemcpy(d_x0, h_x0, size_of_galaxy_array0, hipMemcpyHostToDevice );
    hipMemcpy(d_y0, h_y0, size_of_galaxy_array0, hipMemcpyHostToDevice );
    hipMemcpy(d_z0, h_z0, size_of_galaxy_array0, hipMemcpyHostToDevice );
    hipMemcpy(d_x1, h_x1, size_of_galaxy_array1, hipMemcpyHostToDevice );
    hipMemcpy(d_y1, h_y1, size_of_galaxy_array1, hipMemcpyHostToDevice );
    hipMemcpy(d_z1, h_z1, size_of_galaxy_array1, hipMemcpyHostToDevice );

    int x, y;
    int num_submatrices_x = NUM_GALAXIES0 / SUBMATRIX_SIZE;
    int num_submatrices_y = NUM_GALAXIES1 / SUBMATRIX_SIZE;
    // Take care of edges of matrix.
    if (NUM_GALAXIES0%SUBMATRIX_SIZE != 0)
    {
        num_submatrices_x += 1;
    }
    if (NUM_GALAXIES1%SUBMATRIX_SIZE != 0)
    {
        num_submatrices_y += 1;
    }


    printf("Breaking down the calculations.\n");
    printf("Number of submatrices: %dx%d\n",num_submatrices_x,num_submatrices_y);
    printf("Number of calculations per submatrices: %dx%d\n",SUBMATRIX_SIZE,SUBMATRIX_SIZE);

    int bin_index = 0;
    for(int k = 0; k < num_submatrices_y; k++)
    {
        y = k*SUBMATRIX_SIZE;
        //printf("%d %d\n",k,y);
        for(int j = 0; j < num_submatrices_x; j++)
        {
            x = j*SUBMATRIX_SIZE; 

            //printf("----\n");
            //printf("%d %d\t\t%d %d\n",k,y,j,x);
            //printf("----\n");

            // Set the histogram to all zeros each time.
            hipMemset(dev_hist,0,size_hist_bytes);

            int max_x = NUM_GALAXIES0;
            int max_y = NUM_GALAXIES1;

            distanceMpc<<<grid,block>>>(d_x0, d_y0, d_z0,d_x1, d_y1, d_z1, x, y, max_x, max_y, dev_hist, hist_lower_range, hist_upper_range, nbins, hist_bin_width, log_binning_flag, two_different_files,conv_factor_angle);
            hipMemcpy(hist, dev_hist, size_hist_bytes, hipMemcpyDeviceToHost);

            ////////////////////////////////////////////////////////////////////
            // Sum up the histograms from each thread (hist).
            ////////////////////////////////////////////////////////////////////
            for(int m=0; m<size_hist; m++)
            {
                bin_index = m%(nbins+2);
                hist_array[bin_index] += hist[m];
            }    
        }  
    }

    unsigned long total = 0;

    float lo = hist_lower_range;
    float hi = 0;
    for(int k=0; k<nbins+2; k++)
    {
        if (k==0)
        {
            //fprintf(outfile, "Underflow below %.3e %s %lu \n", lo, ",",  hist_array[k]);
            printf("Underflow below %.3e %s %lu \n", lo, ",",  hist_array[k]);
        }
        else
        {
            if (log_binning_flag==0)
            {
                hi = lo + hist_bin_width;
            }
            else if (log_binning_flag==1)
            {
                //printf("lo: %f\t\tlog(lo): %f\n",lo,log(lo));
                hi = exp(log(lo) + hist_bin_width);
            }
            else if (log_binning_flag==2)
            {
                //printf("lo: %f\t\tlog10(lo): %f\n",lo,log10(lo));
                hi = pow(10,(log10(lo) + hist_bin_width));
            }

            fprintf(outfile, "%.3e %.3e %lu \n",lo,hi,hist_array[k]);
            total += hist_array[k];

            lo = hi;
        }
    }
    printf("total: %lu \n", total);

    fclose(infile0);
    fclose(infile1);
    fclose(outfile);

    free(h_x0);
    free(h_y0);
    free(h_z0);
    free(h_x1);
    free(h_y1);
    free(h_z1);
    free(hist);

    hipFree(d_x0);
    hipFree(d_y0);
    hipFree(d_z0);  
    hipFree(d_x1);
    hipFree(d_y1);
    hipFree(d_z1);  
    hipFree(dev_hist);

    return 0;
}  
//////////////////////////////////////////////////////////////////////





void getDeviceDiagnostics(int tot_gals, int n_coords){

 ////////////////////////////////////////////////////////////////////////////
    // Now get the info from the device.
    ////////////////////////////////////////////////////////////////////////////
   
        printf("\n------ CUDA device diagnostics ------\n\n");

        
        int nx = SUBMATRIX_SIZE;
        int ncalc = nx * nx;
        int gpu_mem_needed = int(tot_gals * sizeof(float)) * n_coords; // need to allocate ra, dec.
        printf("Requirements: %d calculations and %d bytes memory on the GPU \n\n", ncalc, gpu_mem_needed);

        int deviceCount = 0;
        hipError_t error_id = hipGetDeviceCount(&deviceCount);
        if (error_id != hipSuccess) {
            printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
        }
        // This function call returns 0 if there are no CUDA capable devices.
        if (deviceCount == 0)
            printf("There is no device supporting CUDA\n");
        else
            printf("Found %d CUDA Capable device(s)\n", deviceCount);


        int dev=0;
        for (dev = 0; dev < deviceCount; ++dev) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, dev);
            printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

            printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                    (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);


            printf("  Warp size:                                     %d\n", deviceProp.warpSize);
            printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
            printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
                    deviceProp.maxThreadsDim[0],
                    deviceProp.maxThreadsDim[1],
                    deviceProp.maxThreadsDim[2]);
            printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
                    deviceProp.maxGridSize[0],
                    deviceProp.maxGridSize[1],
                    deviceProp.maxGridSize[2]);

            // does this device have enough capcacity for the calculation?
            printf("\n*************\n");

            // check memory
            if((unsigned long long) deviceProp.totalGlobalMem < gpu_mem_needed) printf(" FAILURE: Not eneough memeory on device for this calculation! \n");
            else
            {
                printf("Hurrah! This device has enough memory to perform this calculation\n");

                // check # threads

                int threadsPerBlock = deviceProp.maxThreadsPerBlock; // maximal efficiency exists if we use max # threads per block.
                int blocksPerGrid = int(ceil(ncalc / threadsPerBlock)); // need nx*nx threads total
                if(deviceProp.maxThreadsDim[0] >blocksPerGrid) printf("FAILURE: Not enough threads on the device to do this calculation!\n");
                else
                {
                    printf("Hurrah! This device supports enough threads to do this calculation\n");
                    // how many kernels can we run at once on this machine?
                    int n_mem = floor(deviceProp.totalGlobalMem / float(gpu_mem_needed));
                    int n_threads = floor(threadsPerBlock * deviceProp.maxThreadsDim[0]*deviceProp.maxThreadsDim[1] / float(ncalc) ); // max # threads possible?

                    printf("%d %d  \n",  n_threads, deviceProp.maxThreadsDim[0]);

                    int max_kernels = 0;
                    n_mem<n_threads ? max_kernels = n_mem : max_kernels = n_threads;

                    printf(" you can run %d kernels at a time on this device without overloading the resources \n", max_kernels);
                }
            }

        }

        printf("\n------ End CUDA device diagnostics ------\n\n");
    }
    ////////////////////////////////////////////////////////////////////////////


