#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <math.h>
#include <assert.h>

#include <iostream>
#include <iomanip>
#include <fstream>

#include <unistd.h>



#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include "cutil.h"
using namespace std;

//////////////////////////////////////////////////////////////////////////  
//this version calculates the aperture mass at the galaxy positions. 
////////////////////////////////////////////////////////////////////////// 





void checkCUDAerror(const char *msg);

int checkDeviceSpecs(int number_of_galaxies, int grid_size);


/////////////////////////////////////////////////////////////////////
//   The kernel: calculates the aperture mass, noise and SNR
/////////////////////////////////////////////////////////////////////

__global__ void mApKernel(float* rgamma1, float* rgamma2, float* ra, float* dec, float* mAp_rgamma, float* var_rgamma, float* SN_rgamma,  int tot_gals, float theta_max, int grid_size, float ra_pixsize, float dec_pixsize, float min_ra, float min_dec)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
   
  // need to have the grid coordinates in arcminutes. 
float tempra = idx/grid_size;
float tempdec = idx - grid_size*tempra;
  float this_ra = min_ra + tempra*ra_pixsize;
  float this_dec = min_dec + tempdec*dec_pixsize;
  
  // want to include any tails outside the halo radius to which our filter is tuned......
  int kernel_radius = 1.5*theta_max;
  float ang = 0.0;
  float xc = 0.15; // a constant of the calculation. 
  float x = 0, Q = 0;
  
  float rgammaMap = 0;
  float rgammaVar=0;
  float radiff, decdiff, dist;

  float npoints = 0;

  for(int i=0; i<tot_gals; i++){

    radiff = (float)this_ra-ra[i]; 
    if(abs(radiff)>kernel_radius) continue;
    decdiff=(float)this_dec-dec[i];
    if(abs(decdiff)>kernel_radius || (radiff==0 && decdiff==0)) continue;
    
    dist = sqrtf(radiff*radiff + decdiff*decdiff);
    if(abs(dist)>kernel_radius) continue;
    
    // have to do something a bit complicated for the angle - make sure it's getting the correct range.
    if(radiff==0 && decdiff>0) ang = M_PI/2.0;
    else if(radiff==0 && decdiff<0) ang = -1.0 * M_PI/2.0;
    else if(radiff>0) ang = atanf(decdiff/radiff);
    else if(radiff<0 && decdiff>0) ang = atanf(decdiff/radiff) + M_PI;
    else if(radiff<0 and decdiff<0) ang = atanf(decdiff/radiff)-M_PI;
    
    x = dist / theta_max;
    Q = (1.0 / (1.0 + exp(6.0 - 150.0*x) + exp(-47.0 + 50.0*x))) * (tanh(x/xc) / (x/xc));
    
    rgammaMap+= Q* (-1* (rgamma1[i]*cos(2*ang) + rgamma2[i]*sin(2*ang) ));
    rgammaVar+= Q*Q* (rgamma1[i]*rgamma1[i] + rgamma2[i]*rgamma2[i]);
    
    
     npoints++;

}
  
  // the outputs from this calculation:
  
  mAp_rgamma[idx] = rgammaMap/npoints;// got to normalise by the # gals I did the sum over. 
  var_rgamma[idx] = rgammaVar /(2.0*npoints*npoints); 
  SN_rgamma[idx] = sqrtf(2) * rgammaMap / sqrtf(rgammaVar);
}




////////////////////////////////////////////////////////////////
// setting up the aperture mass calculation
//////////////////////////////////////////////////////////////

int main(int argc, char **argv){
  

  char* input_filename; char* output_filename;
  int number_of_galaxies, grid_size; 
  float filter_rad, min_ra, max_ra, min_dec, max_dec;
  if (argc>1)
    {
      input_filename = argv[1];
      output_filename = argv[2];
      number_of_galaxies = atoi(argv[3]);
      filter_rad = atof(argv[4]);
      grid_size = atoi(argv[5]);
      min_ra = atof(argv[6]);
      max_ra = atof(argv[7]);
      min_dec = atof(argv[8]);
      max_dec = atof(argv[9]);
    }


  int ncalc = grid_size*grid_size;
     
  float ra_pixsize = (max_ra - min_ra)/float(grid_size);
  float dec_pixsize = (max_dec - min_dec)/float(grid_size);

  // CPU memory
  size_t sizeneeded = number_of_galaxies*sizeof(float);
  float *h_rgamma1 = 0;
  float *h_rgamma2 = 0;
  float *h_ra = 0;
  float *h_dec = 0;
  h_rgamma1 = (float*) malloc(sizeneeded);
  h_rgamma2 = (float*) malloc(sizeneeded);
  h_ra = (float*) malloc(sizeneeded);
  h_dec = (float*) malloc(sizeneeded);
  

  ifstream infile;
  infile.open(input_filename);
  
  int i=0;
  float x, y, g1, g2;
  while(1)
    {
      infile>>x>>y>>g1>>g2;
      h_ra[i] = x;
      h_dec[i] = y;
      h_rgamma1[i] = g1;
      h_rgamma2[i] = g2;   
      i += 1;
      if(!infile.good()) break;       

    }
             
    
  // check whether the device has the capacity to do this calculation. 
  // this is taken from the SDK function deviceQuery	
  int max_threads = checkDeviceSpecs(number_of_galaxies, ncalc);

  
  /// first, I need to test whether the device is busy. If so, it can wait a little while.
    while(1){
      size_t testsize = 1*sizeof(float); 
      float *d_test;
      hipMalloc(&d_test, testsize);
      hipError_t err = hipGetLastError();
      if( hipSuccess != err){
	printf("gotta wait for a bit!: %s\n",  hipGetErrorString( err) );
	sleep(10);
      }
      else break;
    }
    
    
    // GPU memory for input 
    float *d_rgamma1, *d_rgamma2, *d_ra, *d_dec;
    hipMalloc(&d_rgamma1, sizeneeded);
    hipMalloc(&d_rgamma2, sizeneeded);
    hipMalloc(&d_ra, sizeneeded);
    hipMalloc(&d_dec, sizeneeded);
    
    // set up vectors for host and device for output. 
    size_t sizeneeded_out = ncalc*sizeof(float);
    float *h_mAp_rgamma,*d_mAp_rgamma, *h_var_rgamma, *d_var_rgamma, *h_SN_rgamma, *d_SN_rgamma;
    
    h_mAp_rgamma = (float*)malloc(sizeneeded_out);
    hipMalloc(&d_mAp_rgamma, sizeneeded_out);
    h_var_rgamma = (float*)malloc(sizeneeded_out);
    hipMalloc(&d_var_rgamma, sizeneeded_out);
    h_SN_rgamma = (float*)malloc(sizeneeded_out);
    hipMalloc(&d_SN_rgamma, sizeneeded_out);
    
    
    //copy vectors from host to device memory
    hipMemcpy(d_rgamma1, h_rgamma1, sizeneeded, hipMemcpyHostToDevice);
    hipMemcpy(d_rgamma2, h_rgamma2, sizeneeded, hipMemcpyHostToDevice);
    hipMemcpy(d_ra, h_ra, sizeneeded, hipMemcpyHostToDevice);
    hipMemcpy(d_dec, h_dec, sizeneeded, hipMemcpyHostToDevice);
    hipMemcpy(d_mAp_rgamma, h_mAp_rgamma, sizeneeded_out, hipMemcpyHostToDevice);
    hipMemcpy(d_var_rgamma, h_var_rgamma, sizeneeded_out, hipMemcpyHostToDevice);
    hipMemcpy(d_SN_rgamma, h_SN_rgamma, sizeneeded_out, hipMemcpyHostToDevice);
    
    //check memory is alright
    if (0==h_rgamma1 || 0==h_rgamma2  || 0==h_ra || 0==h_dec || 0==h_mAp_rgamma || 0==h_var_rgamma || 0==h_SN_rgamma) printf("can't allocate memory on host \n");
    if (0==d_rgamma1 || 0==d_rgamma2  || 0==d_ra || 0==d_dec  || 0==d_mAp_rgamma || 0==d_var_rgamma  || 0==d_SN_rgamma ) printf("can't allocate memory on device \n");
    checkCUDAerror("memory");
    
    
    
    // set up kernel params
    int threadsPerBlock = max_threads; 
    int blocksPerGrid = int(ceil( ncalc / float(max_threads)) ); // need grid_size*grid_size threads total
    printf(" theads per block: %d and blocks per grid: %d for a total of: %d\n", threadsPerBlock, blocksPerGrid, threadsPerBlock*blocksPerGrid);
    
    
    mApKernel<<<blocksPerGrid, threadsPerBlock >>>(d_rgamma1, d_rgamma2, d_ra, d_dec, d_mAp_rgamma,d_var_rgamma, d_SN_rgamma,  number_of_galaxies, filter_rad, grid_size, ra_pixsize, dec_pixsize, min_ra, min_dec);
    checkCUDAerror("kernel");
    
    
    //get the output_mAp back off the device
    hipMemcpy(h_mAp_rgamma, d_mAp_rgamma, sizeneeded_out, hipMemcpyDeviceToHost);
    hipMemcpy(h_var_rgamma, d_var_rgamma, sizeneeded_out, hipMemcpyDeviceToHost);
    hipMemcpy(h_SN_rgamma, d_SN_rgamma, sizeneeded_out, hipMemcpyDeviceToHost);
    
    
    // finally, write out to the output file! 
    
   
    FILE *output_file;
    double sq2=sqrt(2.0);
    output_file = fopen(output_filename, "w");
    fprintf(output_file, " # ra  dec  mAp  Var S/N \n");
    float this_ra, this_dec;
    int tempra, tempdec;

    for(int ii=0 ; ii<ncalc; ii++){

     	tempra = ii/grid_size;	
	tempdec = ii - grid_size*tempra;
   	this_ra = min_ra + tempra*ra_pixsize;
   	this_dec = min_dec + tempdec*dec_pixsize;
	
      fprintf(output_file, "%f %f %f %f %f \n", this_ra, this_dec, h_mAp_rgamma[ii], h_var_rgamma[ii], h_SN_rgamma[ii] ) ;


    }
    fclose(output_file);
    
    printf("successfuly completed!\n");
}





//////////////////////////////////////////////////////////////
//  simple function to check for errors. 
//////////////////////////////////////////////////////////////

void checkCUDAerror(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
	      hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}



///////////////////////////////////////////////////////////////////////////////////////
//  function to check whether GPU device has the specs to perform the calculation. 
//  adapted from cuda SDK deviceQuery example. 
///////////////////////////////////////////////////////////////////////////////////////

int checkDeviceSpecs( int number_of_galaxies, int ncalc){



  int gpu_mem_needed = int(number_of_galaxies * sizeof(float))*4 +  int(ncalc * sizeof(float))*3; // need to allocate gamma1, gamma2, ra, dec and output mAp and var and SN. 
  printf("Requirements: %d calculations and %d bytes memory on the GPU \n\n", ncalc, gpu_mem_needed);  

  int threadsPerBlock=0;
  // now get the info from the device. 
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess) {
    printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
  }
  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0)
    printf("There is no device supporting CUDA\n");
  else
    printf("Found %d CUDA Capable device(s)\n", deviceCount); 
  
  
  int dev, driverVersion = 0;     
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
    
    printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n", 
	   (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
    
    printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);

 // you can uncomment this info if you want to know  bit more about your device specs. 
 //Or just run devicQuery from teh SDK.    
 //   //printf("  Warp size:                                     %d\n", deviceProp.warpSize);
 //   printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
//	   deviceProp.maxThreadsDim[0],
//	   deviceProp.maxThreadsDim[1],
//	   deviceProp.maxThreadsDim[2]);
//    printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
//	   deviceProp.maxGridSize[0],
//	   deviceProp.maxGridSize[1],
//	   deviceProp.maxGridSize[2]);
    
    
    
    // does this device have enough capcacity for the calculation? 
        
    // check memory
    if((unsigned long long) deviceProp.totalGlobalMem < gpu_mem_needed) {
      printf(" FAILURE: Not eneough memeory on device for this calculation! \n");
      exit(1);
    }    
    else
      { 
	printf("Hurrah! This device has enough memory to perform this calculation\n");
	
	// check # threads
	
	threadsPerBlock = deviceProp.maxThreadsPerBlock; // maximal efficiency exists if we use max # threads per block. 
	int blocksPerGrid = int(ceil(ncalc / threadsPerBlock)); // need grid_size*grid_size threads total

	if( int(deviceProp.maxThreadsDim[1])*int(deviceProp.maxThreadsDim[2]) <blocksPerGrid) {
	  printf("FAILURE: Not enough threads on the device to do this calculation!\n");
	    exit(1);
	  }
	else 
	  {
	    printf("Hurrah! This device supports enough threads to do this calculation\n");
	  }
      }

  }// loop over devices
  
  return threadsPerBlock;
}
